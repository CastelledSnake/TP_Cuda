#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define UN_MILLIARD 1000000000

// Compile with command : $nvcc -g --compiler-options -Wall array_addition.cu -o array_addition_par
// With time profiling : $sudo /usr/local/cuda/bin/nvprof <program_name and arguments>

__global__ void random_array(int size, int array[]) {
    /* Generates a random array of integers. into the device memory. */
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	hiprandState state;
	hiprand_init(clock64(), tid, 0, &state);

	for (int i=tid; i<size; i+=blockDim.x * gridDim.x) {
		array[i] = hiprand(&state);
	}
}


__global__ void sum_arrays(int size, int *a1, int *a2, int *result) {
    /* Sums two arrays of integers into the device. */
	for (int i=0; i<size; ++i) {
		result[i] = a1[i] + a2[i];
	}
}


int main(int argc, char *argv[]) {
	int array_size, threads_per_block, nb_blocks;
	//struct timespec t1, t2;
	//long t_a1, t_a2, t_add;
	//long t_add;
	int *d_a1, *d_a2, *d_add;
	int *a1, *a2, *add;
	int max_blocks, max_threads_p_b;
	
	/* Reading all arguments : the size of the arrays to sum, the number of threads per block and the number of blocks to parallelise the execution of the sum.*/
	if (argc != 4) {
		fprintf(stderr, "usage %s array_size threads_per_block nb_blocks\n", argv[0]);
		return EXIT_FAILURE;
	}
	if ((sscanf(argv[1], "%d", &array_size)) == -1) {
		fprintf(stderr, "Impossible to read array_size\n");
		return EXIT_FAILURE;
	}
	if ((sscanf(argv[2], "%d", &threads_per_block)) == -1) {
		fprintf(stderr, "Impossible to read the number of threads per block\n");
		return EXIT_FAILURE;
	}
	if ((sscanf(argv[3], "%d", &nb_blocks)) == -1) {
		fprintf(stderr, "Impossible to read the number of blocks\n");
		return EXIT_FAILURE;
	}
	
	srand(time(NULL));
    // Get capability information about the device.
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	max_blocks = deviceProp.maxThreadsPerMultiProcessor / deviceProp.maxThreadsPerBlock;

	// Check if the number of blocks and threads per block fit with the system's capacities.
	if (nb_blocks > max_blocks) {
		fprintf(stderr, "The device can handle %d blocks at max ; %d given\n", max_blocks, nb_blocks);
		return EXIT_FAILURE;
	}
	max_threads_p_b = deviceProp.maxThreadsPerBlock;
	if (threads_per_block > max_threads_p_b) {
		fprintf(stderr, "The device can handle %d threads per block ; %d given\n", max_threads_p_b, threads_per_block);
		return EXIT_FAILURE;
	}
		
	/* ALLOACTING a1 */
	a1 = (int*)malloc(array_size * sizeof(int));
	//clock_gettime(CLOCK_REALTIME, &t1);
	// Allocate device memory for a1.
	hipMalloc(&d_a1, array_size*sizeof(int));
	// Generate array.
	random_array<<<threads_per_block, nb_blocks>>>(array_size, d_a1);
	//clock_gettime(CLOCK_REALTIME, &t2);
	// Transfer data from device to host for printing.
	/*
	cudaMemcpy(a1, d_a1, array_size*sizeof(int), cudaMemcpyDeviceToHost);
	for (int k=0; k<array_size; k++) {
		fprintf(stdout, "%d ", a1[k]);
	}
	fprintf(stdout, "\n");
	*/
	//t_a1 = (t2.tv_sec - t1.tv_sec) / UN_MILLIARD + (t2.tv_nsec - t1.tv_nsec);
	
	/* ALLOACTING a2 */	
	a2 = (int*) malloc(array_size * sizeof(int));
	//clock_gettime(CLOCK_REALTIME, &t1);
	// Allocate device memory for a2.
	hipMalloc(&d_a2, array_size*sizeof(int));
	// Generate array.
	random_array<<<threads_per_block, nb_blocks>>>(array_size, d_a2);
	//clock_gettime(CLOCK_REALTIME, &t2);
	// Transfer data from host to device memory.
	/*
	cudaMemcpy(a2, d_a2, array_size*sizeof(int), cudaMemcpyDeviceToHost);
	for (int k=0; k<array_size; k++) {
		fprintf(stdout, "%d ", a2[k]);
	}
	fprintf(stdout, "\n");
	*/
	//t_a2 = (t2.tv_sec - t1.tv_sec) / UN_MILLIARD + (t2.tv_nsec - t1.tv_nsec);
	
	/* ALLOACTING add */
	add = (int*) malloc(array_size * sizeof(int));
	// Allocate device memory for add.
	hipMalloc(&d_add, array_size*sizeof(int));
	
	/* COMPUTING ADD */
	//clock_gettime(CLOCK_REALTIME, &t1);
	// The development device can handle 2 blocks and 1024 threads per block simultaneously.
       	//sum_arrays<<<max_blocks, max_threads_p_b>>>(array_size, d_a1, d_a2, d_add);
	sum_arrays<<<threads_per_block, nb_blocks>>>(array_size, d_a1, d_a2, d_add);
	//clock_gettime(CLOCK_REALTIME, &t2);
	//t_add = (t2.tv_sec - t1.tv_sec) / UN_MILLIARD + (t2.tv_nsec - t1.tv_nsec);
	// Transfer data from host to device memory.
	/*	
	cudaMemcpy(add, d_add, array_size * sizeof(int), cudaMemcpyDeviceToHost);
	for (int k=0; k<array_size; k++) {
		fprintf(stdout, "%d ", add[k]);
	}
	fprintf(stdout, "\n ");
	*/

	/* GENERAL CLEANUP */
	hipFree(d_a1);
	hipFree(d_a2);
	hipFree(d_add);
	free(a1);
	free(a2);
	free(add);
	
	//fprintf(stdout, "a1 creation : %ld ns\na2 creation : %ld ns\naddition : %ld ns\n", t_a1, t_a2, t_add);
	//fprintf(stdout, "%ld;%ld;%ld\n", t_a1, t_a2, t_add);
	return 0;
}
